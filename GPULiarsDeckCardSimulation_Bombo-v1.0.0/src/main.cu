#include "util.h"
#include "Simulate.h"

int main(int argc, char* argv[])
{
    if (argc != 2) { std::cerr << "Usage: " << argv[0] << " n\n"; return 1;}

    int numOfSimulations = atoi(argv[1]);

    if (numOfSimulations <= 0) {std::cerr << "Number of Simulations must be positive.\n"; return 1;}

    int* deviceResults, hostResults[POSSIBLE_OUTCOMES] = {0};
    hipMalloc(&deviceResults, POSSIBLE_OUTCOMES * sizeof(int));
    hipMemset(hostResults, 0, POSSIBLE_OUTCOMES * sizeof(int));

    int threadsPerBlock = 256;
    int numOfBlocks = (numOfSimulations + threadsPerBlock - 1) / threadsPerBlock;

    // start timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    simulateCardDraws<<<numOfBlocks, threadsPerBlock>>>(time(NULL), deviceResults, numOfSimulations);

    hipEventRecord(stop);

    hipMemcpy(hostResults, deviceResults, POSSIBLE_OUTCOMES * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(deviceResults);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    for (int i = 0; i < POSSIBLE_OUTCOMES; i++)
    {
        std::cout << i << " valid cards: " << hostResults[i] << std::endl;
    }

    std::cout << "Processing time: " << milliseconds << " ms" << std::endl;

    return 0;
}
