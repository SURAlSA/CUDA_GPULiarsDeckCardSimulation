#include "hip/hip_runtime.h"
#include "Simulate.h"

// variables

const int DECK_SIZE = 20;

const int TOTAL_VALID_CARDS = 8;

const int HAND_SIZE = 5;

const int POSSIBLE_OUTCOMES = HAND_SIZE + 1;

// functions

__global__ void simulateCardDraws(unsigned int seed, int* results, int numOfSimulations)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= numOfSimulations) return;

    // Initialize random state
    hiprandState_t state;
    hiprand_init(seed + tid, 0, 0, &state);

    // Bitmask representation for deck (20 bits)
    unsigned int deck = 0;

    // Efficient placement of valid cards using Fisher-Yates Shuffle
    for (int i = 0; i < TOTAL_VALID_CARDS; i++) 
    {
        int pos = hiprand(&state) % (DECK_SIZE - i);
        for (int j = 0, count = 0; j < DECK_SIZE; j++) 
        {
            if (!(deck & (1 << j))) 
            {
                if (count == pos) 
                {
                    deck |= (1 << j);
                    break;
                }
                count++;
            }
        }
    }

    // Draw HAND_SIZE cards
    int validCount = 0;
    for (int i = 0, count = 0; count < HAND_SIZE; i++) 
    {
        if (hiprand(&state) % (DECK_SIZE - i) < HAND_SIZE - count) 
        {
            validCount += (deck & (1 << i)) ? 1 : 0;
            count++;
        }
    }

    // Store result
    atomicAdd(&results[validCount], 1);
}